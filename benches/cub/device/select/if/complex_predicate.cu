#include "hip/hip_runtime.h"
#include <nvbench/nvbench.cuh>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
// Why is this in detail?
#include <thrust/detail/raw_pointer_cast.h>
#include <cub/device/device_select.cuh>

#include <tbm/range_generator.cuh>

template <typename T, unsigned int OperationsLeft>
struct helper;

template <typename T>
struct helper<T, 1>
{
  static __device__ bool compute(T lhs, T rhs)
  {
    return lhs < rhs;
  }
};

template <typename T, unsigned int OperationsLeft>
struct helper
{
  static __device__ bool compute(T lhs, T rhs)
  {
    return helper<T, OperationsLeft - 1>::compute(std::sqrt(lhs * lhs),
                                                  rhs);
  }
};

template <typename T, unsigned int OperationsCount>
struct complex_select_op
{
  T m_val {};

  explicit complex_select_op(T val)
      : m_val(val)
  {}

  __device__ bool operator()(const T& val)
  {
    return helper<T, OperationsCount>::compute(val, m_val);
  }
};

template <typename T, int OperationsCount, tbm::data_pattern Pattern>
static void basic(nvbench::state &state,
                  nvbench::type_list<T,
                                     nvbench::enum_type<OperationsCount>,
                                     nvbench::enum_type<Pattern>>)
{
  const auto elements = static_cast<int>(state.get_int64("Elements"));

  auto input =
    tbm::make_range_generator<T, tbm::iterator_style::pointer, Pattern>(
      elements);

  thrust::device_vector<T> output(elements);
  thrust::device_vector<T> num_selected(1);

  complex_select_op<T, OperationsCount> select_op{0.42f};

  auto selected_elements =
    thrust::count_if(thrust::device, input.cbegin(), input.cend(), select_op);

  state.add_element_count(elements);
  state.add_global_memory_reads(input.get_allocation_size());
  state.add_global_memory_writes<T>(selected_elements);

  size_t tmp_size;
  hipcub::DeviceSelect::If(nullptr,
                        tmp_size,
                        input.cbegin(),
                        thrust::raw_pointer_cast(output.data()),
                        thrust::raw_pointer_cast(num_selected.data()),
                        elements,
                        select_op);

  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);

  state.exec([&](nvbench::launch &launch) {
    std::size_t temp_size = tmp.size(); // need an lvalue
    hipcub::DeviceSelect::If(thrust::raw_pointer_cast(tmp.data()),
                          temp_size,
                          input.cbegin(),
                          thrust::raw_pointer_cast(output.data()),
                          thrust::raw_pointer_cast(num_selected.data()),
                          elements,
                          select_op);
  });
}

// Column names for type axes:
inline std::vector<std::string> select_if_type_axis_names()
{
  return {"T", "Op", "Pattern"};
}

using types =
  nvbench::type_list<nvbench::float32_t>;

using ops = nvbench::enum_type_list<128>;

using all_input_data_patterns =
  nvbench::enum_type_list<tbm::data_pattern::sequence,
                          tbm::data_pattern::constant,
                          tbm::data_pattern::random>;

NVBENCH_BENCH_TYPES(basic,
                    NVBENCH_TYPE_AXES(types, ops, all_input_data_patterns))
  .set_name("hipcub::DeviceSelect::If")
  .set_type_axes_names(select_if_type_axis_names())
  .add_int64_power_of_two_axis("Elements", nvbench::range(22, 28, 2));
