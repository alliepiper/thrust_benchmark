#include "hip/hip_runtime.h"
#include "thrust/reduce.h"
#include <nvbench/detail/throw.cuh>
#include <nvbench/nvbench.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>

#include <cub/device/device_memcpy.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <cstdint>
#include <limits>
#include <random>
#include <stdexcept>

/**
 * @brief Enum class with options for generating the buffer order within memory
 */
enum class buffer_order
{
  // Buffers are randomly shuffled within memory
  RANDOM,

  // Buffer N+1 resides next to buffer N
  CONSECUTIVE
};

/**
 * @brief Function object class template that takes an offset and returns an
 * iterator at the given offset relative to a fixed base iterator.
 *
 * @tparam IteratorT The random-access iterator type to be returned
 */
template <typename IteratorT>
struct offset_to_ptr_op
{
  template <typename T>
  __host__ __device__ __forceinline__ IteratorT operator()(T offset) const
  {
    return base_it + offset;
  }
  IteratorT base_it;
};

/**
 * @brief Host-side random data generation
 */
template <typename T>
void generate_random_data(
  T *rand_out,
  const std::size_t num_items,
  const T min_rand_val          = std::numeric_limits<T>::min(),
  const T max_rand_val          = std::numeric_limits<T>::max(),
  const std::uint_fast32_t seed = 320981U,
  typename std::enable_if<std::is_integral<T>::value && (sizeof(T) >= 2)>::type
    * = nullptr)
{
  // Initialize random number generator
  std::mt19937 rng(seed);
  std::uniform_int_distribution<T> uni_dist(min_rand_val, max_rand_val);

  // Generate random numbers
  for (std::size_t i = 0; i < num_items; ++i)
  {
    rand_out[i] = uni_dist(rng);
  }
}

/**
 * @brief Used for generating a shuffled but cohesive sequence of output-buffer
 * offsets for the sequence of input-buffers.
 */
template <typename BufferOffsetT,
          typename ByteOffsetT,
          typename BufferSizeItT,
          typename BufferOffsetsOutItT>
void get_shuffled_buffer_offsets(BufferSizeItT buffer_sizes_it,
                                 BufferOffsetT num_buffers,
                                 BufferOffsetsOutItT new_offsets,
                                 const std::uint_fast32_t seed = 320981U)
{
  // We're remapping the i-th buffer to pmt_idxs[i]
  std::mt19937 rng(seed);
  std::vector<BufferOffsetT> pmt_idxs(num_buffers);
  std::iota(pmt_idxs.begin(), pmt_idxs.end(), static_cast<BufferOffsetT>(0));
  std::shuffle(std::begin(pmt_idxs), std::end(pmt_idxs), rng);

  // Compute the offsets using the new mapping
  ByteOffsetT running_offset = {};
  std::vector<ByteOffsetT> permuted_offsets;
  permuted_offsets.reserve(num_buffers);
  for (auto permuted_buffer_idx : pmt_idxs)
  {
    permuted_offsets.emplace_back(running_offset);
    running_offset += buffer_sizes_it[permuted_buffer_idx];
  }

  // Generate the scatter indexes that identify where each buffer was mapped to
  std::vector<BufferOffsetT> scatter_idxs(num_buffers);
  for (BufferOffsetT i = 0; i < num_buffers; i++)
  {
    scatter_idxs[pmt_idxs[i]] = i;
  }

  for (BufferOffsetT i = 0; i < num_buffers; i++)
  {
    new_offsets[i] = permuted_offsets[scatter_idxs[i]];
  }
}

template <typename AtomicT, buffer_order buffer_order>
static void basic(nvbench::state &state,
                  nvbench::type_list<AtomicT, nvbench::enum_type<buffer_order>>)
{
  // Type alias
  using SrcPtrT       = uint8_t *;
  using BufferOffsetT = int32_t;
  using BufferSizeT   = int32_t;
  using ByteOffsetT   = int32_t;

  constexpr auto input_gen  = buffer_order;
  constexpr auto output_gen = buffer_order;

  const auto target_copy_size =
    static_cast<std::size_t>(state.get_int64("Elements"));

  // Make sure buffer ranges are an integer multiple of AtomicT
  const auto min_buffer_size = CUB_ROUND_UP_NEAREST(
    static_cast<std::size_t>(state.get_int64("Min. buffer size")),
    sizeof(AtomicT));
  const auto max_buffer_size = CUB_ROUND_UP_NEAREST(
    static_cast<std::size_t>(state.get_int64("Max. buffer size")),
    sizeof(AtomicT));

  // Skip benchmarks where min. buffer size exceeds max. buffer size
  if (min_buffer_size > max_buffer_size)
  {
    state.skip("Skipping benchmark, as min. buffer size exceeds max. buffer "
               "size.");
    return;
  }

  // Compute number of buffers to generate
  double average_buffer_size = (min_buffer_size + max_buffer_size) / 2.0;
  const auto num_buffers =
    static_cast<std::size_t>(target_copy_size / average_buffer_size);

  // Buffer segment data (their offsets and sizes)
  std::vector<BufferSizeT> h_buffer_sizes(num_buffers);
  std::vector<ByteOffsetT> h_buffer_src_offsets(num_buffers);
  std::vector<ByteOffsetT> h_buffer_dst_offsets(num_buffers);

  // Generate the buffer sizes
  generate_random_data(h_buffer_sizes.data(),
                       h_buffer_sizes.size(),
                       static_cast<BufferSizeT>(min_buffer_size),
                       static_cast<BufferSizeT>(max_buffer_size));

  // Make sure buffer sizes are a multiple of the most granular unit (one
  // AtomicT) being copied (round down)
  for (BufferOffsetT i = 0; i < num_buffers; i++)
  {
    h_buffer_sizes[i] = (h_buffer_sizes[i] / sizeof(AtomicT)) * sizeof(AtomicT);
  }

  if (input_gen == buffer_order::CONSECUTIVE)
  {
    thrust::exclusive_scan(std::cbegin(h_buffer_sizes),
                           std::cend(h_buffer_sizes),
                           std::begin(h_buffer_src_offsets));
  }
  if (output_gen == buffer_order::CONSECUTIVE)
  {
    thrust::exclusive_scan(std::cbegin(h_buffer_sizes),
                           std::cend(h_buffer_sizes),
                           std::begin(h_buffer_dst_offsets));
  }
  // Compute the total bytes to be copied
  ByteOffsetT num_total_bytes = thrust::reduce(std::cbegin(h_buffer_sizes),
                                               std::cend(h_buffer_sizes),
                                               ByteOffsetT{0});

  // Shuffle input buffer source-offsets
  std::uint_fast32_t shuffle_seed = 320981U;
  if (input_gen == buffer_order::RANDOM)
  {
    get_shuffled_buffer_offsets<BufferOffsetT, ByteOffsetT>(
      h_buffer_sizes.data(),
      static_cast<BufferOffsetT>(h_buffer_sizes.size()),
      h_buffer_src_offsets.data(),
      shuffle_seed);
    shuffle_seed += 42;
  }

  // Shuffle input buffer source-offsets
  if (output_gen == buffer_order::RANDOM)
  {
    get_shuffled_buffer_offsets<BufferOffsetT, ByteOffsetT>(
      h_buffer_sizes.data(),
      static_cast<BufferOffsetT>(h_buffer_sizes.size()),
      h_buffer_dst_offsets.data(),
      shuffle_seed);
  }

  // Get temporary storage requirements
  size_t temp_storage_bytes = 0;
  CubDebugExit(hipcub::DeviceMemcpy::Batched(nullptr,
                                          temp_storage_bytes,
                                          static_cast<SrcPtrT *>(nullptr),
                                          static_cast<SrcPtrT *>(nullptr),
                                          static_cast<BufferSizeT *>(nullptr),
                                          num_buffers));

  // Compute total device memory requirements
  std::size_t total_required_mem = num_total_bytes +                     //
                                   num_total_bytes +                     //
                                   (num_buffers * sizeof(ByteOffsetT)) + //
                                   (num_buffers * sizeof(ByteOffsetT)) + //
                                   (num_buffers * sizeof(BufferSizeT)) + //
                                   temp_storage_bytes;                   //

  // Get available device memory
  std::size_t available_device_mem =
    state.get_device().has_value()
      ? state.get_device().value().get_global_memory_usage().bytes_free
      : 0;

  // Skip benchmark there's insufficient device memory available
  if (available_device_mem < total_required_mem)
  {
    state.skip("Skipping benchmark due to insufficient device memory");
    return;
  }

  thrust::device_vector<uint8_t> d_temp_storage(temp_storage_bytes);

  // Add benchmark reads
  state.add_element_count(num_total_bytes);
  state.add_global_memory_reads<char>(num_total_bytes, "data");
  state.add_global_memory_reads<ByteOffsetT>(num_buffers, "buffer src offsets");
  state.add_global_memory_reads<ByteOffsetT>(num_buffers, "buffer dst offsets");
  state.add_global_memory_reads<BufferSizeT>(num_buffers, "buffer sizes");

  // Add benchmark writes
  state.add_global_memory_writes<char>(num_total_bytes, "data");

  // Prepare random data segment (which serves for the buffer sources)
  thrust::device_vector<uint8_t> d_in_buffer(num_total_bytes);
  thrust::device_vector<uint8_t> d_out_buffer(num_total_bytes);

  // Populate the data source buffer
  thrust::fill(std::begin(d_in_buffer),
               std::end(d_in_buffer),
               std::numeric_limits<uint8_t>::max());

  // Raw pointers into the source and destination buffer
  auto d_in  = thrust::raw_pointer_cast(d_in_buffer.data());
  auto d_out = thrust::raw_pointer_cast(d_out_buffer.data());

  // Prepare device-side data
  thrust::device_vector<ByteOffsetT> d_buffer_src_offsets =
    h_buffer_src_offsets;
  thrust::device_vector<ByteOffsetT> d_buffer_dst_offsets =
    h_buffer_dst_offsets;
  thrust::device_vector<BufferSizeT> d_buffer_sizes = h_buffer_sizes;

  // Prepare d_buffer_srcs
  offset_to_ptr_op<SrcPtrT> src_transform_op{static_cast<SrcPtrT>(d_in)};
  hipcub::TransformInputIterator<SrcPtrT, offset_to_ptr_op<SrcPtrT>, ByteOffsetT *>
    d_buffer_srcs(thrust::raw_pointer_cast(d_buffer_src_offsets.data()),
                  src_transform_op);

  // Prepare d_buffer_dsts
  offset_to_ptr_op<SrcPtrT> dst_transform_op{static_cast<SrcPtrT>(d_out)};
  hipcub::TransformInputIterator<SrcPtrT, offset_to_ptr_op<SrcPtrT>, ByteOffsetT *>
    d_buffer_dsts(thrust::raw_pointer_cast(d_buffer_dst_offsets.data()),
                  dst_transform_op);

  state.exec([&](nvbench::launch &launch) {
    std::size_t temp_size = d_temp_storage.size(); // need an lvalue
    hipcub::DeviceMemcpy::Batched(thrust::raw_pointer_cast(d_temp_storage.data()),
                               temp_size,
                               d_buffer_srcs,
                               d_buffer_dsts,
                               thrust::raw_pointer_cast(d_buffer_sizes.data()),
                               num_buffers,
                               launch.get_stream());
  });
}

// Column names for type axes:
inline std::vector<std::string> type_axis_names()
{
  return {"AtomicT", "Buffer Order"};
}

// Benchmark for unaligned buffers and buffers aligned to four bytes
using atomic_type = nvbench::type_list<nvbench::uint8_t, nvbench::uint32_t>;

using buffer_orders =
  nvbench::enum_type_list<buffer_order::RANDOM, buffer_order::CONSECUTIVE>;

NVBENCH_DECLARE_ENUM_TYPE_STRINGS(
  buffer_order,
  [](buffer_order data_gen_mode) {
    switch (data_gen_mode)
    {
      case buffer_order::RANDOM:
        return "Random";
      case buffer_order::CONSECUTIVE:
        return "Consecutive";
      default:
        break;
    }
    NVBENCH_THROW(std::runtime_error, "{}", "Unknown data_pattern");
  },
  [](buffer_order data_gen_mode) {
    switch (data_gen_mode)
    {
      case buffer_order::RANDOM:
        return "Buffers are randomly shuffled within memory";
      case buffer_order::CONSECUTIVE:
        return "Consecutive buffers reside cohesively in memory";
      default:
        break;
    }
    NVBENCH_THROW(std::runtime_error, "{}", "Unknown data_pattern");
  })

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(atomic_type, buffer_orders))
  .set_name("hipcub::DeviceMemcpy::Batched")
  .set_type_axes_names(type_axis_names())
  .add_int64_axis("Min. buffer size", {1, 64 * 1024})
  .add_int64_axis("Max. buffer size", {8, 64, 256, 1024, 64 * 1024})
  .add_int64_power_of_two_axis("Elements", nvbench::range(25, 29, 2));
