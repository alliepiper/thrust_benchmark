#include <nvbench/nvbench.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <thrust/detail/raw_pointer_cast.h>

#include <cub/device/device_scan.cuh>

using value_types = nvbench::type_list<nvbench::int32_t, nvbench::float32_t>;

template <typename InputType, typename OutputType>
void mixed_types(nvbench::state &state,
                 nvbench::type_list<InputType, OutputType>)
{
  const auto size = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<InputType> input(size);
  thrust::device_vector<OutputType> output(size);

  thrust::sequence(input.begin(), input.end());

  state.add_global_memory_reads<InputType>(size, "InputSize");
  state.add_global_memory_writes<OutputType>(size, "OutputSize");
  state.add_element_count(size);

  size_t tmp_size;
  hipcub::DeviceScan::InclusiveSum(nullptr,
                                tmp_size,
                                input.cbegin(),
                                output.begin(),
                                static_cast<int>(input.size()));
  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);

  state.exec([&input, &output, &tmp](nvbench::launch &launch) {
    std::size_t temp_size = tmp.size(); // need an lvalue
    hipcub::DeviceScan::InclusiveSum(thrust::raw_pointer_cast(tmp.data()),
                                  temp_size,
                                  input.cbegin(),
                                  output.begin(),
                                  static_cast<int>(input.size()),
                                  launch.get_stream());
  });
}
using value_types = nvbench::type_list<nvbench::int32_t, nvbench::float32_t>;
template <typename T>
void mixed_types(nvbench::state &state, nvbench::type_list<T, T>)
{
  state.skip("Types are not mixed.");
}
NVBENCH_BENCH_TYPES(mixed_types, NVBENCH_TYPE_AXES(value_types, value_types))
  .set_name("hipcub::DeviceScan::InclusiveSum (mixed types)")
  .set_type_axes_names({"In", "Out"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 30, 2))
  .set_timeout(2)
  .set_skip_time(100e-6 /* us */);
