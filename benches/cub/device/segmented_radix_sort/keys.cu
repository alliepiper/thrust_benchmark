#include "segments_generator.cuh"
#include "type_lists.cuh"

#include <nvbench/nvbench.cuh>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>

#include <cub/device/device_segmented_radix_sort.cuh>

#include <type_traits>

template <typename T, sort_direction SortDirection>
void basic(nvbench::state &state,
           nvbench::type_list<T, nvbench::enum_type<SortDirection>>)
{
  const int elements = static_cast<int>(state.get_int64("Elements"));

  thrust::device_vector<T> input(elements);
  thrust::device_vector<T> output(elements);

  const auto pattern = state.get_string("Pattern");
  const auto offsets = gen_offsets(pattern, elements);

  const auto bits = state.get_string("Bits");

  // hipcub::DeviceSegmentedRadixSort reads data multiple times. Limiting the
  // number of bits is a way of having accurate throughput estimation.
  const int first_bit = 0;
  const int last_bit  = bits == "all" ? sizeof(T) * 8 : 4;

  const int num_segments = static_cast<int>(offsets.size() - 1);

  const T *d_input                   = thrust::raw_pointer_cast(input.data());
  T *d_output                        = thrust::raw_pointer_cast(output.data());
  const nvbench::uint32_t *d_offsets = thrust::raw_pointer_cast(offsets.data());

  std::size_t temp_storage_bytes{};
  if constexpr (SortDirection == sort_direction::ascending)
  {
    hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr,
                                            temp_storage_bytes,
                                            d_input,
                                            d_output,
                                            elements,
                                            num_segments,
                                            d_offsets,
                                            d_offsets + 1,
                                            first_bit,
                                            last_bit);
  }
  else
  {
    hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr,
                                                      temp_storage_bytes,
                                                      d_input,
                                                      d_output,
                                                      elements,
                                                      num_segments,
                                                      d_offsets,
                                                      d_offsets + 1,
                                                      first_bit,
                                                      last_bit);
  }

  thrust::device_vector<nvbench::uint8_t> temp_storage(temp_storage_bytes);
  nvbench::uint8_t *d_temp_storage =
    thrust::raw_pointer_cast(temp_storage.data());

  state.add_element_count(elements);
  state.add_global_memory_reads<nvbench::uint32_t>(num_segments, "Segments");
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);

  state.exec([&](nvbench::launch &launch) {
    if constexpr (SortDirection == sort_direction::ascending)
    {
      hipcub::DeviceSegmentedRadixSort::SortKeys(d_temp_storage,
                                              temp_storage_bytes,
                                              d_input,
                                              d_output,
                                              elements,
                                              num_segments,
                                              d_offsets,
                                              d_offsets + 1,
                                              first_bit,
                                              last_bit,
                                              launch.get_stream());
    }
    else
    {
      hipcub::DeviceSegmentedRadixSort::SortKeysDescending(d_temp_storage,
                                                        temp_storage_bytes,
                                                        d_input,
                                                        d_output,
                                                        elements,
                                                        num_segments,
                                                        d_offsets,
                                                        d_offsets + 1,
                                                        first_bit,
                                                        last_bit,
                                                        launch.get_stream());
    }
  });
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types, directions))
  .set_name("hipcub::DeviceSegmentedRadixSort::SortKeys")
  .add_int64_power_of_two_axis("Elements", nvbench::range(20, 30, 2))
  .add_string_axis("Bits", {"few", "all"})
  .add_string_axis("Pattern", {"small", "large", "random"});
