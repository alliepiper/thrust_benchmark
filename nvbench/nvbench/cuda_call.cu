#include <nvbench/cuda_call.cuh>

#include <stdexcept>
#include <string>

#include <fmt/format.h>

namespace nvbench
{

namespace cuda_call
{

void throw_error(const std::string &filename,
                 std::size_t lineno,
                 const std::string &command,
                 hipError_t error_code)
{
  throw std::runtime_error(fmt::format("{}:{}: Cuda API call returned error: "
                                       "{}: {}\nCommand: '{}'",
                                       filename,
                                       lineno,
                                       hipGetErrorName(error_code),
                                       hipGetErrorString(error_code),
                                       command));
}

} // namespace cuda_call

} // namespace nvbench
