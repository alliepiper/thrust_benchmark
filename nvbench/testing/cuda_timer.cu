#include <nvbench/cuda_timer.cuh>

#include <nvbench/hip_stream.cuh>
#include <nvbench/types.cuh>

#include "sleep_kernel.cuh"
#include "test_asserts.cuh"

#include <fmt/format.h>

void test_basic(hipStream_t time_stream,
                hipStream_t exec_stream,
                bool expected)
{
  nvbench::cuda_timer timer;

  NVBENCH_CUDA_CALL(hipDeviceSynchronize());

  timer.start(time_stream);
  sleep_kernel<<<1, 1, 0, exec_stream>>>(0.25);
  timer.stop(time_stream);

  NVBENCH_CUDA_CALL(hipDeviceSynchronize());
  const bool captured = timer.get_duration() > 0.25;
  ASSERT_MSG(captured == expected,
             "Unexpected result from timer: {} seconds (expected {})",
             timer.get_duration(),
             (expected ? "> 0.25s" : "< 0.25s"));
}

void test_basic()
{
  nvbench::hip_stream stream1;
  nvbench::hip_stream stream2;

  test_basic(stream1, stream1, true);
  test_basic(stream1, stream2, false);
}

int main() { test_basic(); }
